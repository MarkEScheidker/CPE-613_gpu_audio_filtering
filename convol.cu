#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <string>
#include <stdlib.h>
#include "fdacoefs.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__constant__ float const_conv_kernel[BL];

__global__ void tiled_convolution_1D_kernel(int16_t *result, const int16_t *audio_data, int data_size, int kernel_size) {
    extern __shared__ int16_t shared_data[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int t_idx = threadIdx.x;
    int half_kernel = kernel_size / 2;

    if (t_idx < half_kernel) {
        int left_index = idx - half_kernel;
        shared_data[t_idx] = (left_index < 0) ? 0 : audio_data[left_index];
    }

    shared_data[t_idx + half_kernel] = (idx < data_size) ? audio_data[idx] : 0;

    if (t_idx < half_kernel) {
        int right_index = idx + blockDim.x;
        shared_data[t_idx + blockDim.x + half_kernel] = (right_index >= data_size) ? 0 : audio_data[right_index];
    }
    __syncthreads();

    float value = 0.0f;
    for (int j = 0; j < kernel_size; ++j) {
        int shared_index = t_idx + j;
        value += (shared_data[shared_index] / 32768.0f) * const_conv_kernel[j];
    }

    if (idx < data_size) {
        value = value * 32768.0f;
        value = fmaxf(-32768.0f, fminf(32767.0f, roundf(value)));
        result[idx] = static_cast<int16_t>(value);
    }
}


struct header {
    unsigned int ChunkID;
    unsigned int ChunkSize;
    unsigned int Format;
    unsigned int Subchunk1ID;
    unsigned int Subchunk1Size;
    unsigned short AudioFormat;
    unsigned short NumChannels;
    unsigned int SampleRate;
    unsigned int ByteRate;
    unsigned short BlockAlign;
    unsigned short BitsPerSample;
    unsigned int Subchunk2ID;
    unsigned int Subchunk2Size;
};

int main() {
    std::string infile_name = "white_noise.wav";
    std::string outfile_name = "output.wav";

    //open both files
    std::ifstream infile(infile_name, std::ios::binary);
    std::ofstream outfile(outfile_name, std::ios::binary);

    //quit if files cannot be opened
    if (!infile) {
      std::cout << "Error: Could not open input file." << std::endl;
      return 1;
    }
    if (!outfile) {
      std::cout << "Error: Could not open output file." << std::endl;
      return 1;
    }

    //create header struct and fill with data from input file, checking for supported sample rate before continuing
    header my_header;
    infile.read(reinterpret_cast<char*>(&my_header), sizeof(header));

    //get number of samples
    int num_samples = my_header.Subchunk2Size/2;

    //allocate the area required to store the data
    int16_t* input_audio_data = (int16_t*) malloc(num_samples * sizeof(int16_t));
    int16_t *output_audio_data = (int16_t *)malloc(num_samples * sizeof(int16_t));

    //get data from the file and store it in memory
    infile.read(reinterpret_cast<char*>(input_audio_data), num_samples * sizeof(int16_t));

    //allocate gpu memory
    int16_t *gpu_audio_data, *gpu_result;
    float *gpu_convol_kernel;
    checkCudaErrors(hipMalloc(&gpu_audio_data, num_samples*sizeof(int16_t)));
    checkCudaErrors(hipMalloc(&gpu_convol_kernel, BL*sizeof(float)));
    checkCudaErrors(hipMalloc(&gpu_result, num_samples*sizeof(int16_t)));

    //copy data to gpu memory
    checkCudaErrors(
        hipMemcpy(
            gpu_audio_data, 
            input_audio_data, 
            num_samples*sizeof(int16_t), 
            hipMemcpyHostToDevice
        )
    );

    checkCudaErrors(
        hipMemcpy(
            gpu_convol_kernel, 
            B, 
            BL*sizeof(float), 
            hipMemcpyHostToDevice
        )
    );

    // copy filter to global memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_conv_kernel), B, BL * sizeof(float));
    
    float duration_ms = 0.0f;
    hipEvent_t start, stop;

    // timing code instantiation
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    //kernel launch parameters
    int kernel_size = BL;
    int blockSize = 256;
    int numBlocks = (num_samples + blockSize - 1) / blockSize;
    int sharedMemSize = (blockSize + 2 * (kernel_size / 2)) * sizeof(int16_t);

    //launch the kernel
    checkCudaErrors(hipEventRecord(start));

    // Kernel launch
    tiled_convolution_1D_kernel<<<numBlocks, blockSize, sharedMemSize>>>(gpu_result, gpu_audio_data, num_samples, kernel_size);

    // get the time
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&duration_ms, start, stop));

    //copy result back to host
    checkCudaErrors(hipMemcpy(output_audio_data, gpu_result, num_samples * sizeof(int16_t), hipMemcpyDeviceToHost));

    printf("naive time: %.10fms\n", duration_ms);

    //write the header to the output file
    outfile.write(reinterpret_cast<const char*>(&my_header), sizeof(header));
    //write the audio data
    outfile.write(reinterpret_cast<const char*>(output_audio_data), num_samples * sizeof(int16_t));

    //free memory
    free(input_audio_data);
    free(output_audio_data);
    checkCudaErrors(hipFree(gpu_audio_data));
    checkCudaErrors(hipFree(gpu_convol_kernel));
    checkCudaErrors(hipFree(gpu_result));
    infile.close();
    outfile.close();
    return 0;

}